/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "common.h"
#include "utility.h"

namespace tensorrt_llm
{
namespace kernels
{
template <WeightOnlyQuantType QType, typename WeightOnlyFlag, template <typename T> class ActOp, bool Zero, bool Bias,
    int N_PER_BLOCK, int BATCH, int BLOCK_SIZE>
struct WeightOnlyBatchedGemvKernelLauncher
{
    static void run(const WeightOnlyParams& params, hipStream_t stream);
};

template <WeightOnlyQuantType QType, typename WeightOnlyFlag, template <typename T> class ActOp, int N_PER_BLOCK,
    int BATCH, int BLOCK_SIZE>
void select_zero_bias(const WeightOnlyParams& params, hipStream_t stream)
{
    if (params.zeros && params.bias)
    {
        WeightOnlyBatchedGemvKernelLauncher<QType, WeightOnlyFlag, ActOp, true, true, N_PER_BLOCK, BATCH,
            BLOCK_SIZE>::run(params, stream);
    }
    else if (params.zeros && !params.bias)
    {
        WeightOnlyBatchedGemvKernelLauncher<QType, WeightOnlyFlag, ActOp, true, false, N_PER_BLOCK, BATCH,
            BLOCK_SIZE>::run(params, stream);
    }
    else if (!params.zeros && params.bias)
    {
        WeightOnlyBatchedGemvKernelLauncher<QType, WeightOnlyFlag, ActOp, false, true, N_PER_BLOCK, BATCH,
            BLOCK_SIZE>::run(params, stream);
    }
    else
    {
        WeightOnlyBatchedGemvKernelLauncher<QType, WeightOnlyFlag, ActOp, false, false, N_PER_BLOCK, BATCH,
            BLOCK_SIZE>::run(params, stream);
    }
}

template <WeightOnlyQuantType QType, typename WeightOnlyFlag, int N_PER_BLOCK, int BATCH, int BLOCK_SIZE>
void select_activation(const WeightOnlyParams& params, hipStream_t stream)
{
    switch (params.act_func_type)
    {
        // Currently, activation function is not called in the plugin
#if 0
    case WeightOnlyActivationFunctionType::Gelu:
    {
        select_zero_bias<QType, WeightOnlyFlag, GeluActivation, N_PER_BLOCK, BATCH, BLOCK_SIZE>(params, stream);
        break;
    }
    case WeightOnlyActivationFunctionType::Relu:
    {
        select_zero_bias<QType, WeightOnlyFlag, ReluActivation, N_PER_BLOCK, BATCH, BLOCK_SIZE>(params, stream);
        break;
    }
#endif
    case WeightOnlyActivationFunctionType::Identity:
    {
        select_zero_bias<QType, WeightOnlyFlag, IdentityActivation, N_PER_BLOCK, BATCH, BLOCK_SIZE>(params, stream);
        break;
    }
    default:
    {
        throw std::runtime_error("Use unsupported activation");
        break;
    }
    }
}

template <typename WeightOnlyFlag, int N_PER_BLOCK, int BATCH, int BLOCK_SIZE>
void select_quant_type(const WeightOnlyParams& params, hipStream_t stream)
{
    if (params.quant_type == WeightOnlyQuantType::Int4b)
    {
        select_activation<WeightOnlyQuantType::Int4b, WeightOnlyFlag, N_PER_BLOCK, BATCH, BLOCK_SIZE>(params, stream);
    }
    else if (params.quant_type == WeightOnlyQuantType::Int8b)
    {
        select_activation<WeightOnlyQuantType::Int8b, WeightOnlyFlag, N_PER_BLOCK, BATCH, BLOCK_SIZE>(params, stream);
    }
    else
    {
        throw std::runtime_error("Unknown QuantType");
    }
}

template <int N_PER_BLOCK, int BATCH, int BLOCK_SIZE>
void select_groupwise_weight_only(const WeightOnlyParams& params, hipStream_t stream)
{
    if (params.weight_only_type == WeightOnlyType::GroupWise && params.group_size == 64)
    {
        select_quant_type<WeightOnlyGroupWise<64>, N_PER_BLOCK, BATCH, BLOCK_SIZE>(params, stream);
    }
    else if (params.weight_only_type == WeightOnlyType::GroupWise && params.group_size == 128)
    {
        select_quant_type<WeightOnlyGroupWise<128>, N_PER_BLOCK, BATCH, BLOCK_SIZE>(params, stream);
    }
    else
    {
        throw std::runtime_error("Only support groupwise weight only for gs=64/128");
    }
}

void weight_only_batched_gemv_launcher(const WeightOnlyParams& params, hipStream_t stream)
{
    assert(params.act_func_type == WeightOnlyActivationFunctionType::Identity);
    assert(params.weight_only_type == WeightOnlyType::GroupWise
        || (params.weight_only_type == WeightOnlyType::PerChannel && params.bias == nullptr
            && params.zeros == nullptr));
    if (params.weight_only_type == WeightOnlyType::PerChannel)
    {
        if (params.quant_type == WeightOnlyQuantType::Int4b)
        {
            switch (params.m)
            {
            case 1:
            {
                WeightOnlyBatchedGemvKernelLauncher<WeightOnlyQuantType::Int4b, WeightOnlyPerChannel,
                    IdentityActivation, false, false, 1, 1, 192>::run(params, stream);
                break;
            }
            case 2:
            {
                WeightOnlyBatchedGemvKernelLauncher<WeightOnlyQuantType::Int4b, WeightOnlyPerChannel,
                    IdentityActivation, false, false, 2, 2, 128>::run(params, stream);
                break;
            }
            case 3:
            {
                WeightOnlyBatchedGemvKernelLauncher<WeightOnlyQuantType::Int4b, WeightOnlyPerChannel,
                    IdentityActivation, false, false, 2, 3, 256>::run(params, stream);
                break;
            }
            case 4:
            {
                WeightOnlyBatchedGemvKernelLauncher<WeightOnlyQuantType::Int4b, WeightOnlyPerChannel,
                    IdentityActivation, false, false, 4, 4, 256>::run(params, stream);
                break;
            }
            default:
            {
                throw std::runtime_error("Weight only cuda kernel only supported bs <= 4");
                break;
            }
            }
        }
        else if (params.quant_type == WeightOnlyQuantType::Int8b)
        {
            switch (params.m)
            {
            case 1:
            {
                WeightOnlyBatchedGemvKernelLauncher<WeightOnlyQuantType::Int8b, WeightOnlyPerChannel,
                    IdentityActivation, false, false, 2, 1, 256>::run(params, stream);
                break;
            }
            case 2:
            {
                WeightOnlyBatchedGemvKernelLauncher<WeightOnlyQuantType::Int8b, WeightOnlyPerChannel,
                    IdentityActivation, false, false, 2, 2, 256>::run(params, stream);
                break;
            }
            case 3:
            {
                WeightOnlyBatchedGemvKernelLauncher<WeightOnlyQuantType::Int8b, WeightOnlyPerChannel,
                    IdentityActivation, false, false, 2, 3, 256>::run(params, stream);
                break;
            }
            case 4:
            {
                WeightOnlyBatchedGemvKernelLauncher<WeightOnlyQuantType::Int8b, WeightOnlyPerChannel,
                    IdentityActivation, false, false, 2, 4, 256>::run(params, stream);
                break;
            }
            default:
            {
                throw std::runtime_error("Weight only cuda kernel only supported bs <= 4");
                break;
            }
            }
        }
    }
    else if (params.weight_only_type == WeightOnlyType::GroupWise)
    {
        switch (params.m)
        {
        case 1:
        {
            select_groupwise_weight_only<2, 1, 256>(params, stream);
            break;
        }
        case 2:
        {
            select_groupwise_weight_only<2, 2, 256>(params, stream);
            break;
        }
        case 3:
        {
            select_groupwise_weight_only<2, 3, 128>(params, stream);
            break;
        }
        case 4:
        {
            select_groupwise_weight_only<2, 4, 128>(params, stream);
            break;
        }
        default:
        {
            throw std::runtime_error("Weight only cuda kernel only supported bs <= 4");
            break;
        }
        }
    }
}
} // namespace kernels
} // namespace tensorrt_llm
