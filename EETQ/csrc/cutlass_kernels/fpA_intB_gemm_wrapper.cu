#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include "hipcub/hipcub.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <c10/cuda/CUDAGuard.h>
#include "fpA_intB_gemm_wrapper.h"
#include "fpA_intB_gemm.h"
#include "cutlass_preprocessors.h"
#include "cuda_utils.h"
#include "weightOnlyBatchedGemv/enabled.h"
#include "weightOnlyBatchedGemv/kernelLauncher.h"
#include "torch_utils.h"

#include <vector>

namespace ft = fastertransformer;

int getWorkspaceSize(const int m, const int n, const int k)
{
    // These are the min tile sizes for each config, which would launch the maximum number of blocks
    const int max_grid_m = (m + 31) / 32;
    const int max_grid_n = (n + 127) / 128;
    const int split_k_limit = 7;
    // We need 4 bytes per block in the worst case. We launch split_k_limit in z dim.
    return max_grid_m * max_grid_n * split_k_limit * 4;
}

std::vector<torch::Tensor>
symmetric_quantize_last_axis_of_tensor(torch::Tensor &weight,
                                       py::object &quant_type,
                                       bool return_unprocessed_quantized_tensor)
{
    CHECK_CPU(weight);
    CHECK_CONTIGUOUS(weight);
    TORCH_CHECK(weight.numel() != 0, "weight should not be empty tensor");
    TORCH_CHECK(weight.dim() == 2 || weight.dim() == 3, "Invalid dim. The dim of weight should be 2 or 3");

    torch::ScalarType _quant_type = torch::python::detail::py_object_to_dtype(quant_type);
    auto _st = weight.scalar_type();
    TORCH_CHECK(_st == torch::kFloat32 || _st == torch::kFloat16, "Invalid datatype. Weight must be FP16 or FP32");
    TORCH_CHECK(_quant_type == torch::kInt8 || _quant_type == at::ScalarType::QUInt4x2, "Must be int4 or int8 quantization");
    ft::QuantType ft_quant_type = ft::get_ft_quant_type(_quant_type);

    const size_t num_experts = weight.dim() == 2 ? 1 : weight.size(0);
    const size_t num_rows    = weight.size(-2);
    const size_t num_cols    = weight.size(-1);

    const size_t bits_in_type      = ft::get_bits_in_quant_type(ft_quant_type);
    const size_t bytes_per_out_col = num_cols * bits_in_type / 8;

    const size_t input_mat_size     = num_rows * num_cols;
    const size_t quantized_mat_size = num_rows * bytes_per_out_col;

    std::vector<long int> quantized_weight_shape;
    std::vector<long int> scale_shape;
    if (weight.dim() == 2) {
        quantized_weight_shape = {long(num_rows), long(bytes_per_out_col)};
        scale_shape            = {long(num_cols)};
    }
    else if (weight.dim() == 3) {
        quantized_weight_shape = {long(num_experts), long(num_rows), long(bytes_per_out_col)};
        scale_shape            = {long(num_experts), long(num_cols)};
    }
    else {
        TORCH_CHECK(false, "Invalid weight dimension. Weight must have dim 2 or 3");
    }

    torch::Tensor unprocessed_quantized_weight =
        torch::empty(quantized_weight_shape, torch::dtype(torch::kInt8).device(torch::kCPU).requires_grad(false));

    torch::Tensor processed_quantized_weight = torch::empty_like(unprocessed_quantized_weight);

    torch::Tensor scales = torch::empty(scale_shape, torch::dtype(weight.dtype()).device(torch::kCPU).requires_grad(false));

    int8_t *unprocessed_quantized_weight_ptr = reinterpret_cast<int8_t *>(unprocessed_quantized_weight.data_ptr());
    int8_t *processed_quantized_weight_ptr = reinterpret_cast<int8_t *>(processed_quantized_weight.data_ptr());

    if (weight.scalar_type() == at::ScalarType::Float)
    {
        ft::symmetric_quantize<float, float>(processed_quantized_weight_ptr,
                                             unprocessed_quantized_weight_ptr,
                                             reinterpret_cast<float *>(scales.data_ptr()),
                                             reinterpret_cast<const float *>(weight.data_ptr()),
                                             {num_rows, num_cols},
                                             ft_quant_type);
    }
    else if (weight.scalar_type() == at::ScalarType::Half)
    {
        ft::symmetric_quantize<half, half>(processed_quantized_weight_ptr,
                                           unprocessed_quantized_weight_ptr,
                                           reinterpret_cast<half *>(scales.data_ptr()),
                                           reinterpret_cast<const half *>(weight.data_ptr()),
                                           {num_rows, num_cols},
                                           ft_quant_type);
    }
    else
    {
        TORCH_CHECK(false, "Invalid data type. Weight must be FP32/FP16");
    }

    if (return_unprocessed_quantized_tensor)
    {
        return std::vector<torch::Tensor>{unprocessed_quantized_weight, processed_quantized_weight, scales};
    }

    return std::vector<torch::Tensor>{processed_quantized_weight, scales};
}

torch::Tensor preprocess_weights_cuda(torch::Tensor &origin_weight,
                                      bool is_int4)
{
    // guarantee the weight is cpu tensor
    CHECK_CPU(origin_weight);

    torch::Tensor preprocessed_quantized_weight = torch::empty_like(origin_weight);
    int8_t *preprocessed_quantized_weight_ptr = reinterpret_cast<int8_t *>(preprocessed_quantized_weight.data_ptr());
    const int8_t *row_major_quantized_weight_ptr = reinterpret_cast<const int8_t *>(origin_weight.data_ptr());
    size_t rows = origin_weight.size(-2);
    size_t cols = origin_weight.size(-1);
    int arch = ft::getSMVersion();
    ft::preprocess_weights(preprocessed_quantized_weight_ptr,
                                          row_major_quantized_weight_ptr,
                                          rows,
                                          cols,
                                          is_int4,
                                          arch);
    return preprocessed_quantized_weight;
}

torch::Tensor w8_a16_gemm_forward_cuda(torch::Tensor &input,
                                       torch::Tensor &weight,
                                       torch::Tensor &scale)
{
    c10::cuda::CUDAGuard device_guard(input.device());
    // TORCH_CHECK(input.dim() == 3 || input.dim() == 2, "Invalid input dim: ", input.dim());
    const int m = input.dim() == 2 ? input.size(0) : input.size(0) * input.size(1);
    const int k = input.size(-1);
    const int n = weight.size(-1);
    auto options = torch::TensorOptions().dtype(input.dtype()).device(input.device());
    torch::Tensor output = input.dim() == 2 ? torch::empty({m, n}, options) : torch::empty({input.size(0), input.size(1), n}, options);
    const ft::half *input_ptr = reinterpret_cast<ft::half *>(input.data_ptr());
    const uint8_t *weight_ptr = reinterpret_cast<const uint8_t *>(weight.data_ptr());
    const ft::half *scale_ptr = reinterpret_cast<ft::half *>(scale.data_ptr());
    ft::half *output_ptr = reinterpret_cast<ft::half *>(output.data_ptr());
    // const int max_size = std::max(n, k);
    // size_t workspace_size = getWorkspaceSize(m, max_size, max_size);
    // void *ptr = nullptr;
    // char *workspace_ptr = workspace_size > 0 ? (char *)hipMalloc((void **)&ptr, workspace_size) : nullptr;
    const bool use_cuda_kernel = m <= SMALL_M_FAST_PATH;
    // const bool use_cuda_kernel = false; 
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    if(use_cuda_kernel){
        tensorrt_llm::kernels::WeightOnlyActivationType weight_only_act_type = tensorrt_llm::kernels::WeightOnlyActivationType::FP16;
        tensorrt_llm::kernels::WeightOnlyQuantType weight_only_quant_type = tensorrt_llm::kernels::WeightOnlyQuantType::Int8b;
        tensorrt_llm::kernels::WeightOnlyParams params{weight_ptr, reinterpret_cast<const uint8_t *>(scale.data_ptr()), nullptr,
            reinterpret_cast<half *>(input.data_ptr()), nullptr, nullptr, reinterpret_cast<half *>(output.data_ptr()), m, n, k, 0, weight_only_quant_type,
            tensorrt_llm::kernels::WeightOnlyType::PerChannel,
            tensorrt_llm::kernels::WeightOnlyActivationFunctionType::Identity, weight_only_act_type};
        tensorrt_llm::kernels::weight_only_batched_gemv_launcher(params, stream);
    }
    else
        ft::gemm_fp16_int(
            input_ptr,
            weight_ptr,
            scale_ptr,
            output_ptr,
            m, n, k,
            nullptr,
            0,
            stream);
    return output;
}


torch::Tensor w8_a16_gemm_forward_cuda_(torch::Tensor &input,
                                        torch::Tensor &weight,
                                        torch::Tensor &scale,
                                        torch::Tensor &output,
                                        const int m,
                                        const int n,
                                        const int k)
{
    c10::cuda::CUDAGuard device_guard(input.device());

    const ft::half *input_ptr = reinterpret_cast<ft::half *>(input.data_ptr());
    const uint8_t *weight_ptr = reinterpret_cast<const uint8_t *>(weight.data_ptr());
    const ft::half *scale_ptr = reinterpret_cast<ft::half *>(scale.data_ptr());
    ft::half *output_ptr = reinterpret_cast<ft::half *>(output.data_ptr());
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    ft::gemm_fp16_int(
        input_ptr,
        weight_ptr,
        scale_ptr,
        output_ptr,
        m, n, k,
        nullptr,
        0,
        stream);
    return output;
}