/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "kernel.h"

namespace tensorrt_llm
{
namespace kernels
{

template struct WeightOnlyBatchedGemvKernelLauncher<WeightOnlyQuantType::Int4b, WeightOnlyPerChannel,
    IdentityActivation, false, false, 1, 1, 192>;

template struct WeightOnlyBatchedGemvKernelLauncher<WeightOnlyQuantType::Int4b, WeightOnlyGroupWise<64>,
    IdentityActivation, true, true, 2, 1, 256>;
template struct WeightOnlyBatchedGemvKernelLauncher<WeightOnlyQuantType::Int4b, WeightOnlyGroupWise<64>,
    IdentityActivation, true, false, 2, 1, 256>;
template struct WeightOnlyBatchedGemvKernelLauncher<WeightOnlyQuantType::Int4b, WeightOnlyGroupWise<64>,
    IdentityActivation, false, true, 2, 1, 256>;
template struct WeightOnlyBatchedGemvKernelLauncher<WeightOnlyQuantType::Int4b, WeightOnlyGroupWise<64>,
    IdentityActivation, false, false, 2, 1, 256>;

template struct WeightOnlyBatchedGemvKernelLauncher<WeightOnlyQuantType::Int4b, WeightOnlyGroupWise<128>,
    IdentityActivation, true, true, 2, 1, 256>;
template struct WeightOnlyBatchedGemvKernelLauncher<WeightOnlyQuantType::Int4b, WeightOnlyGroupWise<128>,
    IdentityActivation, true, false, 2, 1, 256>;
template struct WeightOnlyBatchedGemvKernelLauncher<WeightOnlyQuantType::Int4b, WeightOnlyGroupWise<128>,
    IdentityActivation, false, true, 2, 1, 256>;
template struct WeightOnlyBatchedGemvKernelLauncher<WeightOnlyQuantType::Int4b, WeightOnlyGroupWise<128>,
    IdentityActivation, false, false, 2, 1, 256>;

} // namespace kernels
} // namespace tensorrt_llm
