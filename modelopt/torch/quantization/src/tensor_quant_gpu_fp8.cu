#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023 NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved. SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include <ATen/ATen.h>
#include <hip/hip_fp8.h>
#include <torch/extension.h>

#define BLOCK_SIZE 128

#define AT_DISPATCH_CASE_FLOATING_TYPES(...)                                                       \
  AT_DISPATCH_CASE(at::ScalarType::Double, __VA_ARGS__)                                            \
  AT_DISPATCH_CASE(at::ScalarType::Float, __VA_ARGS__)                                             \
  AT_DISPATCH_CASE(at::ScalarType::Half, __VA_ARGS__)                                              \
  AT_DISPATCH_CASE(at::ScalarType::BFloat16, __VA_ARGS__)

#define AT_DISPATCH_FLOATING_TYPES(TYPE, NAME, ...)                                                \
  AT_DISPATCH_SWITCH(TYPE, NAME, AT_DISPATCH_CASE_FLOATING_TYPES(__VA_ARGS__))

template <typename T> __global__ void fake_e4m3fy_kernel(const T *inputs, size_t n, T *outputs) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int idx = 4 * tid; idx < 4 * (tid + 1) && idx < n; ++idx) {
    outputs[idx] = static_cast<T>(
        static_cast<float>(static_cast<__hip_fp8_e4m3_fnuz>(static_cast<float>(inputs[idx]))));
  }
}

at::Tensor fake_e4m3fy_cuda(at::Tensor inputs) {
  size_t numel = inputs.numel();
  auto outputs = torch::empty_like(inputs);
  AT_DISPATCH_FLOATING_TYPES(inputs.type().scalarType(), "fake_e4m3fy_cuda", [&] {
    fake_e4m3fy_kernel<<<numel / (BLOCK_SIZE * 4) + 1, BLOCK_SIZE>>>(
        inputs.data_ptr<scalar_t>(), numel, outputs.data_ptr<scalar_t>());
  });
  return outputs;
}
